#include "hip/hip_runtime.h"
﻿
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include "..//..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define SIZE 10

//int a{3};

void Fill_Matrix_CPU(__int32 ** Matrix);
void Fill_Matrix_CPU(__int32 Matrix[][SIZE]);
void Show_Matrix_CPU(__int32 ** Matrix);
void Show_Matrix_CPU(__int32 Matrix[][SIZE]);

__global__ void Multiply_Matrix(__int32* Matrix);

int main(int argc, char* argv[])
{
	//int a{4};

	//_STD cout << ::a << NEW_LINE;

	/*
		ALLOCATING NEEDED MEMORY ON CPU (exactly in RAM)
	*/
	__int32** Matrix_CPU;
	//__int32 Matrix_CPU[SIZE][SIZE]{};
	Matrix_CPU = (__int32**)malloc(SIZE * sizeof(__int32*));

	for (size_t i = 0; i < SIZE; ++i)
	{
		*(Matrix_CPU + i) = (__int32*)malloc(SIZE * sizeof(__int32));
		//Matrix_CPU[i] = (__int32*)malloc(SIZE * sizeof(__int32));
	}
	////////////////////////////////////////////////////

	//FILL MATRIX WITH RANDOM VALUES
	Fill_Matrix_CPU(Matrix_CPU);

	//_STD cout << Matrix_CPU[0] << NEW_LINE;
	//_STD cout << Matrix_CPU[1] << NEW_LINE;
	//_STD cin.get();
	//SHOW FILLED UP ARRAY
	Show_Matrix_CPU(Matrix_CPU);

//	_STD cout << sizeof(__int32) << ' ' << /*sizeof(char*) or sizeof(__int32*)  <<*/ NEW_LINE;


	/*
		ALLOCATING NEEDED MEMORY ON GPU
	*/
	__int32* Matrix_GPU;
	HANDLE_ERROR(hipMalloc((void**)&Matrix_GPU, (SIZE * SIZE) *  sizeof(__int32)));	//GPU interprets 2D array as a flat array !

	////////////////////////////////////////////////////


	//COPY CPU ARRAY TO GPU
	//HANDLE_ERROR(hipMemcpy(Matrix_GPU, Matrix_CPU, (SIZE * SIZE) * sizeof(__int32), HostToDevice));	//only for statics array from HOST's!

	for (size_t i = 0; i < SIZE; ++i)
	{
		hipMemcpy(Matrix_GPU + i * SIZE, *(Matrix_CPU + i), sizeof(__int32) * SIZE, HostToDevice);
	}

	////////////////////////////////////////////////////

	Multiply_Matrix<<<SIZE,SIZE>>>(Matrix_GPU);


	//COPY FROM GPU TO CPU
	//HANDLE_ERROR(hipMemcpy(Matrix_CPU, Matrix_GPU, (SIZE * SIZE) * sizeof(__int32), DeviceToHost));	//only for statics array

	for (size_t i = 0; i < SIZE; ++i)
	{
		hipMemcpy(*(Matrix_CPU + i), Matrix_GPU + i * SIZE, sizeof(__int32) * SIZE, DeviceToHost);	//for dynamic allocation
	}
	////////////////////////////////////////////////////


	//SHOW RESULTS
	Show_Matrix_CPU(Matrix_CPU);

	/*
		FREEING PREVIOUSLY ALOCATED MEMORY
	*/
	//ON CPU
	for (size_t i = 0; i < SIZE; ++i)
	{
		free(*(Matrix_CPU + i));
	}
	free(Matrix_CPU);

	////ON GPU
	/*for (size_t i = 0; i < SIZE; ++i)
	{
		HANDLE_ERROR(hipFree(*(Matrix_CPU + i)));
	}*/
	HANDLE_ERROR(hipFree(Matrix_GPU));
	////////////////////////////////////////////////////


	system("pause");
	return 0;
}

void Fill_Matrix_CPU(__int32** Matrix)
{
	__int32 value = 0;
	for (size_t i = 0; i < SIZE; ++i)
	{
		for (size_t j = 0; j < SIZE; ++j)
		{
			*(*(Matrix + i) + j) = value++;
			//Matrix[i][j] = NULL;
		}
	}
}

void Fill_Matrix_CPU(__int32 Matrix[][SIZE])
{
	__int32 value = 0;
	for (size_t i = 0; i < SIZE; ++i)
	{
		for (size_t j = 0; j < SIZE; ++j)
		{
			//*(*(Matrix + i) + j) = value++;
			Matrix[i][j] = 1;
		}
	}
}

void Show_Matrix_CPU(__int32** Matrix)
{
	for (size_t i = 0; i < SIZE; ++i)
	{
		_STD cout << '|' << ' ';
		for (size_t j = 0; j < SIZE; ++j)
		{
			_STD cout << *(*(Matrix + i) + j) << ' ';
		}
		_STD cout << '|' << NEW_LINE;
	}
	_STD cout << NEW_LINE;
}

void Show_Matrix_CPU(__int32 Matrix[][SIZE])
{
	for (size_t i = 0; i < SIZE; ++i)
	{
		_STD cout << '|' << ' ';
		for (size_t j = 0; j < SIZE; ++j)
		{
			_STD cout << *(*(Matrix + i) + j) << ' ';
		}
		_STD cout << '|' << NEW_LINE;
	}
	_STD cout << NEW_LINE;
}

__global__ void Multiply_Matrix(__int32* Matrix)
{
	int id_x = threadIdx.x + blockIdx.x * blockDim.x;
	int id_y = threadIdx.y + blockIdx.y * blockDim.y;

	while (id_x < SIZE)
	{
		while (id_y < SIZE)
		{
			//printf("A[%d][%d]\n", id_y, id_x);
			//printf("A[%d][%d]\n", id_y * SIZE + id_x);
			//Matrix[id_y+ id_x] *= 2;
			Matrix[id_y * SIZE + id_x] *= 2;
			id_y += blockDim.y * gridDim.y;
			//printf("\n");
		}
		id_x += blockDim.x * gridDim.x;
	}
}