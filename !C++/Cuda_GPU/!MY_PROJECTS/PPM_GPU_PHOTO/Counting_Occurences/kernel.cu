#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/device_functions.h"


#include "../..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <random>
#include <chrono>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define N 100

#define BlocksPerGrid (N/10)
#define ThreadsPerBlock (N/10)

////////////////////////////////////////////////////////
/*
	RANDOM GENERATOR
*/

auto start = _STD chrono::system_clock::now();
_STD random_device random_engine;							
_STD mt19937 generator(random_engine());

////////////////////////////////////////////////////////


void Fill_Array(__int32* array, const size_t& size, const short int choice);
void Show_Array(__int32* array, const size_t& size);
void Show_Array_and_Occurences(const __int32* array_integer, const __int32* array_occurences, const size_t& size);

/*
	GPU FUNCTIONS
*/
__global__ void Count_Occurences(const __int32 * array_integer, __int32* array_occurences, const size_t& size);
////////////////////////////////////////////////////////

int main(int argc, char* argv[])
{
	size_t array_size = N; //or self amount

	/*
		CPU ARRAYS
	*/
	__int32* integer_array{};
	__int32* integer_array_occurences{};
	integer_array = (__int32*)malloc(array_size * sizeof(__int32));
	integer_array_occurences = (__int32*)malloc(array_size * sizeof(__int32));
	////////////////////////////////////////////////////////

	/*
		GPU ARRAYS
	*/
	__int32* integer_array_GPU{};
	__int32* integer_array_GPU_occurences{};
	HANDLE_ERROR(hipMalloc((void**)& integer_array_GPU, array_size * sizeof(__int32)));
	HANDLE_ERROR(hipMalloc((void**)& integer_array_GPU_occurences, array_size * sizeof(__int32)));
	////////////////////////////////////////////////////////


	Fill_Array(integer_array, array_size, 3);
	//Show_Array(integer_array, array_size);


	HANDLE_ERROR(hipMemcpy(integer_array_GPU, integer_array, array_size * sizeof(__int32), HostToDevice));

	//Counting the occurence of the numbers in these arrays
	Count_Occurences <<< BlocksPerGrid, ThreadsPerBlock >>> (integer_array_GPU, integer_array_GPU_occurences, array_size);


	HANDLE_ERROR(hipMemcpy(integer_array_occurences, integer_array_GPU_occurences, array_size * sizeof(__int32), DeviceToHost));



	Show_Array_and_Occurences(integer_array, integer_array_occurences, array_size);

	/*
		Free the memory
	*/
	//ON CPU
	free(integer_array);
	free(integer_array_occurences);

	//ON GPU
	hipFree(integer_array_GPU);
	hipFree(integer_array_GPU_occurences);
	////////////////////////////////////////////////////////


	system("pause");
	return 0;
}


/*
	ARRAY FILLER
	1 -> For self assignment
	2 -> For random engine assignment
	3 -> For basic null values, currently it is a 0
*/
void Fill_Array(__int32* array, const size_t& size, const short int choice)
{
	if (choice == 1)
	{
		for (size_t i = 0; i < size; ++i)
		{
			array[i] = i;
		}
	}
	else if (choice == 2)
	{
		std::uniform_int_distribution<> dis(1, size);			//range of our randomly generator
		for (size_t i = 0; i < size; ++i)
		{
			array[i] = dis(generator);
		}
	}
	else if (choice == 3)
	{
		for (size_t i = 0; i < size; ++i)
		{
			array[i] = 0;
		}
	}
	else
	{
		_STD cerr << "ERROR:: [WRONG NUMBER OF CHOICE] " << NEW_LINE;
	}
}

void Show_Array(__int32* array, const size_t& size)
{
	for (size_t i = 0; i < size; ++i)
	{
		_STD cout << array[i] << ' ';
	}
	_STD cout << NEW_LINE;
}

void Show_Array_and_Occurences(const __int32* array_integer, const __int32* array_occurences, const size_t& size)
{
	for (size_t i = 0; i < size; ++i)
	{
		_STD cout << array_integer[i] << " -> " << array_occurences[i] << NEW_LINE;
	}
	_STD cout << NEW_LINE;
}


//GPU FUNCTIONS

__global__ void Count_Occurences(const __int32* array_integer, __int32* array_occurences, const size_t& size)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t occurence{};
	__shared__ size_t cache_array_of_occurences[ThreadsPerBlock];//change it into array of occurented already
	int cache_index = threadIdx.x;

	while (id < N)
	{
		int id_j = id + 1;
		while (id_j < N)
		{
			if (array_integer[id] == array_integer[id_j])
			{
				++occurence;
			}
			id_j += 1;
		}
		id += blockDim.x * gridDim.x;
	}

	cache_array_of_occurences[cache_index] = occurence;

	//very important to put below this function __syncthreads cause
	//we have to know when all of the threads will end their work!!!
	__syncthreads();

	array_occurences[threadIdx.x + blockIdx.x * blockDim.x] = cache_array_of_occurences[cache_index];
	
	__syncthreads();
}