#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/device_functions.h"


#include "..//..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>



////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define N 100
#define Amount_Of_Blocks 655
#define Amount_Of_Threads 1024

class Pixel_GPU
{
private:
	/*
		PRIVATE VARIABLES
	*/
	unsigned char R;
	unsigned char G;
	unsigned char B;
	int color_range;
	//////////////////////////////////////////////////////////////////////////////
protected:
	/*
		PROTECTED VARIABLES
	*/

	//////////////////////////////////////////////////////////////////////////////
public:
	/*
		KONSTRUKTORY
	*/
	__host__ __device__ Pixel_GPU();
	__host__ __device__ explicit Pixel_GPU(const unsigned char R, const unsigned char G, const unsigned char B);
	__host__ __device__ Pixel_GPU(const Pixel_GPU& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		FUNKCJE PUBLIC
	*/
	__host__ void Show_Object() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		SETTERY
	*/
	__host__ __device__ void Set_R(const unsigned char R);
	__host__ __device__ void Set_G(const unsigned char G);
	__host__ __device__ void Set_B(const unsigned char B);
	__host__ __device__ void Set_Color_Range(const int Color_Range);
	//////////////////////////////////////////////////////////////////////////////
	/*
		GETTERY
	*/
	__host__ __device__ const unsigned char Get_R() const;
	__host__ __device__ const unsigned char Get_G() const;
	__host__ __device__ const unsigned char Get_B() const;
	__host__ __device__ const int Get_Color_Range() const;
	//__host__ __device__ Pixel_GPU * Get_Pointer() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		OPERATORY
	*/
	//JEDNOARGUMENTOWE
	__host__ __device__ Pixel_GPU& operator=(const Pixel_GPU& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		DESTRUKTOR
	*/
	virtual ~Pixel_GPU();
	//////////////////////////////////////////////////////////////////////////////
};

Pixel_GPU* Host_Array{};
size_t global_size{};
__int64 unique{};
__int32 cos_CPU[Amount_Of_Blocks]{};

void Fill_Array(const _STD string& file_path);
void Show_Array(Pixel_GPU* Pixel_array, const size_t& size);
__global__ void Show_Array_GPU(Pixel_GPU* Pixel_array, const size_t * size);
__global__ void Counting_Unique_Colors(Pixel_GPU * Pixel_array, __int64 * unique_colors, const size_t * size);
__global__ void Counting_Unique_Colors(Pixel_GPU * Pixel_array, bool * Is_Unique_or_Not , const size_t * size);
__global__ void Increase(__int64 *& counter);
__global__ void Show_Device_Variables(size_t * size);

/////////////////////////////////////////////////////////////////////
__global__ void Foo_GPU(__int32 * array);
void Foo_CPU(__int32 * array);
/////////////////////////////////////////////////////////////////////

int main(int argc, char* argv[])
{
	//__int64 * unique_colors{};
	//__int64 unique{};

	//size_t * size{};

	//HANDLE_ERROR(hipMalloc((void**)& unique_colors, sizeof(__int64)));
	//HANDLE_ERROR(hipMalloc((void**)& size, sizeof(size_t)));

	////unique_colors = 0;
	////unique = 0;
	////size = 0;

	//Fill_Array("Lena.ppm");

	////Show_Array(Host_Array, global_size);

	//Pixel_GPU* Device_Array{};
	//hipMalloc((void**)& Device_Array, global_size * sizeof(Pixel_GPU));


	//hipMemcpy(Device_Array, Host_Array, global_size * sizeof(Pixel_GPU), HostToDevice);
	////hipMemcpy(&global_size, &size, sizeof(size_t), HostToDevice);
	//hipMemcpy(size, &global_size, sizeof(size_t), HostToDevice);
	////size = global_size;

	//Counting_Unique_Colors <<<655,1024>>> (Device_Array, unique_colors, size);
	////Counting_Unique_Colors <<<1,1024>>> (Device_Array, unique_colors, size);



	//hipMemcpy(Host_Array, Device_Array, global_size * sizeof(Pixel_GPU), DeviceToHost);
	//Show_Array(Host_Array, global_size);

	//hipMemcpy(&unique, unique_colors, sizeof(__int64), DeviceToHost);
	////unique = unique_colors;
	//_STD cout << unique << NEW_LINE;

	//hipFree(unique_colors);
	//hipFree(size);
	//hipFree(Device_Array);

	//free(Host_Array);

	

	/*__int64 *counter_CPU;
	__int64 *counter_GPU;

	counter_CPU = (__int64*)malloc(sizeof(__int64));
	HANDLE_ERROR(hipMalloc((void**)&counter_GPU, sizeof(__int64)));


	Increase <<<10, 10 >>> (counter_GPU);

	HANDLE_ERROR(hipMemcpy(counter_CPU, counter_GPU, sizeof(__int64) , DeviceToHost));


	_STD cout << *counter_CPU << NEW_LINE;


	HANDLE_ERROR(hipFree(counter_GPU));
	free(counter_CPU);*/



	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////


	Fill_Array("Lena.ppm");


	///////////////////////////////////////////////////////////////
	Pixel_GPU* Device_Array{};
	hipMalloc((void**)& Device_Array, global_size * sizeof(Pixel_GPU));

	bool* Is_Unique_or_Not{};
	hipMalloc((void**)& Is_Unique_or_Not, global_size * sizeof(bool));
	///////////////////////////////////////////////////////////////


	///////////////////////////////////////////////////////////////
	size_t* size{};
	hipMalloc((void**)& size, sizeof(size_t));
	hipMemset(size, 0, sizeof(size_t));
	///////////////////////////////////////////////////////////////


	///////////////////////////////////////////////////////////////
	__int64* unique_colors{};
	hipMalloc((void**)& unique_colors, sizeof(__int64));
	hipMemset(unique_colors, 0, sizeof(__int64));
	///////////////////////////////////////////////////////////////


	///////////////////////////////////////////////////////////////
	/*__int32* cos_GPU{};
	hipMalloc((void**)& cos_GPU, sizeof(__int32) * Amount_Of_Blocks);
	hipMemset(cos_GPU, 0, sizeof(__int32) * Amount_Of_Blocks);*/
	///////////////////////////////////////////////////////////////


	hipMemcpy(Device_Array, Host_Array, global_size * sizeof(Pixel_GPU), HostToDevice);
	hipMemcpy(size, &global_size, sizeof(size_t), HostToDevice);


	//Case for tests only
	//_STD cout << global_size << NEW_LINE;
	//Show_Device_Variables <<<1, 1>>>(size);
	//hipMemcpy(&global_size, size, sizeof(size_t), DeviceToHost);
	//_STD cout << global_size << NEW_LINE;
	///////////////////////////////////////////////////////////////


	Counting_Unique_Colors <<<Amount_Of_Blocks, Amount_Of_Threads>>> (Device_Array, Is_Unique_or_Not, size);

	//hipMemcpy(Host_Array, Device_Array, global_size * sizeof(Pixel_GPU), DeviceToHost);
	hipMemcpy(Host_Array, Device_Array, sizeof(Pixel_GPU)* global_size, DeviceToHost);
	//for (size_t i = 0; i < global_size; ++i)
	//{
	//	hipMemcpy(Host_Array, Device_Array , sizeof(Pixel_GPU) * global_size, DeviceToHost);	//for dynamic allocation
	//}

	//hipMemcpy(&unique, unique_colors, sizeof(__int64), DeviceToHost);
	//for (size_t i = 0; i < global_size; ++i)
	//{
	//	if()
	//}

	//_STD cout << "Unique colors: " << unique << NEW_LINE;

	//for (size_t i = 0; i < global_size; ++i)
	//{
	//	if (Host_Array->Get_Color_Range() == 255)
	//	{
	//		unique++;
	//	}
	//}
	Show_Array_GPU <<< 1,1 >>> (Device_Array, size);
	/*Show_Array(Host_Array, global_size);
	*///_STD cout << "Unique colors: " << unique << NEW_LINE;

	//Case for tests only
	/*Foo_GPU <<< 50, 1 >>> (cos_GPU);
	hipMemcpy(cos_CPU, cos_GPU, sizeof(__int32)* Amount_Of_Blocks, DeviceToHost);
	Foo_CPU(cos_CPU);*/
	///////////////////////////////////////////////////////////////\




	//Delete the allocated memory
	hipFree(size);
	hipFree(Device_Array);
	hipFree(Is_Unique_or_Not);
	hipFree(unique_colors);

	free(Host_Array);

	system("pause");
	return 0;
}


void Fill_Array(const _STD string& file_path)
{
	_STD fstream file;
	file.open(file_path.c_str(), std::ios_base::in);
	_STD string line{};
	__int32 a{ 1 };
	Pixel_GPU Temporary_Pixel{};
	std::string extension{};					//extension (p1,p2,p3 etc)
	__int32 width{};							//width
	__int32 height{};							//height
	__int32 color_range{};						//color range
	static size_t i = 0;
	if (file.good() == false)
	{
		exit(0);
	}
	else
	{
		while (file.eof() == false)
		{
			if (a < 5)
			{
				_STD getline(file, line);
				if (a == 1)
				{
					extension = line;
				}
				else if (a == 3)
				{
					_STD string line_second = "";
					int counter = 0;
					for (int i = 0; i < line.size(); i++)
					{
						if (line[i] == ' ')
						{
							width = stoi(line_second);
							line_second = "";
						}
						else
						{
							line_second += line[i];
						}
						counter++;
					}
					height = stoi(line_second);
					global_size = static_cast<size_t>(width * height);
					Host_Array = (Pixel_GPU*)malloc(global_size * sizeof(Pixel_GPU));
				}
				else if (a == 4)
				{
					color_range = (stoi(line));
				}
			}
			else if (a >= 5)
			{
				int color = 0;
				file >> color;
				Temporary_Pixel.Set_R(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_G(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_B(color);
				//Temporary_Pixel.Set_Color_Range(color_range);
				color = 0;
				Host_Array[i++] = Temporary_Pixel;
			}
			line = "";
			a++;
		}
	}
	_STD cout << "--------------------------" << NEW_LINE;
	_STD cout << "  Information" << NEW_LINE;
	_STD cout << "  Loaded format: " << extension << NEW_LINE;
	_STD cout << "  Width: " << width << NEW_LINE;
	_STD cout << "  Height: " << height << NEW_LINE;
	_STD cout << "  Color Range: " << color_range << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	//_STD cout << "  Array size: " << static_cast<size_t>(Pixel_array.size()) << NEW_LINE;
	_STD cout << "  Array size: " << static_cast<size_t>((global_size)) << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	file.close();
}

void Show_Array(Pixel_GPU* Pixel_array, const size_t& size)
{
	for (size_t i = 0; i < size; i++)
	{
		if (Pixel_array[i].Get_Color_Range() > 255)
		{
			Pixel_array[i].Show_Object();
			_STD cin.get();
		}
	/*	Pixel_array[i].Show_Object();
		_STD cin.get();*/
	}
}

__global__ void Show_Array_GPU(Pixel_GPU* Pixel_array, const size_t* size)
{
	int a{};
	for (size_t i = 0; i < *size; i++)
	{
		if (Pixel_array[i].Get_Color_Range() == 255)
		{
			printf("The Size is following: %u \n", Pixel_array[i].Get_Color_Range());
			//++a;
			//_STD cin.get();
		}
	}
	//printf("The Size is following: %u \n", a);
}

__global__ void Counting_Unique_Colors(Pixel_GPU* Pixel_array, __int64 * unique_colors, const size_t * size)
{
	//int id = threadIdx.x + blockIdx.x * blockDim.x;
	//int j = threadIdx.y + blockIdx.y * blockDim.y;
	//bool is_unique = true;
	//const int color_range = Pixel_array[0].Get_Color_Range();
	//if (id < *size)
	//{
	//	//if (Pixel_array[id].Get_Color_Range() > color_range)
	//	//{
	//	//	continue;
	//	//}
	//	//else
	//	//{
	//	
	//	//}
	//	//__syncthreads();
	//	while (j < *size)
	//	{
	//		/*	Pixel_array[j].Set_Color_Range(static_cast<int>(color_range + 100));
	//			Pixel_array[j].Set_R(static_cast<int>(color_range + 100));*/
	//	/*	if (Pixel_array[id].Get_R() == Pixel_array[j].Get_R() && Pixel_array[id].Get_G() == Pixel_array[j].Get_G() && Pixel_array[id].Get_B() == Pixel_array[j].Get_B())
	//		{
	//			Pixel_array[j].Set_Color_Range(static_cast<int>(color_range + 100));
	//		}*/
	//		Pixel_array[j].Set_Color_Range(static_cast<int>(color_range + 100));
	//		j++;
	//	}
	//	//__syncthreads();
	//	//Pixel_array[id].Set_Color_Range(static_cast<int>(color_range + 100));
	//	//__syncthreads();
	//	/*Pixel_array[id].Set_Color_Range(static_cast<int>(color_range + 100));
	//	Pixel_array[id].Set_R(static_cast<int>(color_range + 100));
	//	id += blockDim.x * gridDim.x;*/
	//}
	//int id_x = threadIdx.x + blockIdx.x * blockDim.x;
	int id_x = blockIdx.x;
	//int id_y = threadIdx.y + blockIdx.y * blockDim.y;
	//const int color_range = Pixel_array[0].Get_Color_Range();

	printf("The Size is following: %u [%u] \n", *size);
	if (id_x < *size)
	{
		/*while (id_y < *size)
		{
			if (Pixel_array[id_x].Get_R() == Pixel_array[id_y * (*size) + id_x].Get_R() && Pixel_array[id_x].Get_G() == Pixel_array[id_y * (*size) + id_x].Get_G() && Pixel_array[id_x].Get_B() == Pixel_array[id_y * (*size) + id_x].Get_B())
			{
				Pixel_array[id_x].Set_Color_Range(static_cast<int>(color_range + 100));
				__syncthreads();
			}
			id_y += blockDim.y * gridDim.y;
		}

		if (Pixel_array[id_x].Get_Color_Range() == color_range)
		{
			++(*unique_colors);
			__syncthreads();
		}*/
		++(*unique_colors);
		//__syncthreads();
		//id_x += blockDim.x * gridDim.x;
	}
	
	//(*unique_colors) = 1000;
	//printf("Unique colors: %u \n", *unique_colors);
}

__global__ void Counting_Unique_Colors(Pixel_GPU* Pixel_array, bool * Is_Unique_or_Not, const size_t* size)
{
	int id_x = threadIdx.x + blockIdx.x * blockDim.x;
	int id_y = threadIdx.y + blockIdx.y * blockDim.y;
	const int color_range = Pixel_array[0].Get_Color_Range();

	//printf("The Size is following: %u [%u] \n", *size);
	while (id_x < *size)
	{
		if (Pixel_array[id_x].Get_Color_Range() > 255)
		{
			continue;
		}
		else
		{
			while ((id_y + 1) < *size)
			{
				if (Pixel_array[id_x].Get_R() == Pixel_array[id_y * (*size) + id_x].Get_R() && 
					Pixel_array[id_x].Get_G() == Pixel_array[id_y * (*size) + id_x].Get_G() && 
					Pixel_array[id_x].Get_B() == Pixel_array[id_y * (*size) + id_x].Get_B())
				{
					Pixel_array[id_x].Set_Color_Range(static_cast<int>(color_range + (id_y * (*size) + id_x) + 1));
					printf("Color range: %u \n", Pixel_array[id_x].Get_Color_Range());
					//break;
					__syncthreads();
				}
				id_y += blockDim.y * gridDim.y;
			}
		}
		
		//__syncthreads();
		id_x += blockDim.x * gridDim.x;
	}
	//__syncthreads();

	//if (Pixel_array[id_x].Get_Color_Range() == color_range)
	//{
	//
	//	__syncthreads();
	//}
}

__global__ void Increase(__int64 *& counter)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < N)
	{
		__syncthreads();
		++((counter));
		__syncthreads();
	}
}

__global__ void Show_Device_Variables(size_t * size)
{
	printf("The Size is following: %u \n", *size);
	//*size = 100;
}

__global__ void Foo_GPU(__int32* array)
{
	int id = blockIdx.x;
	//int id = threadIdx.x + blockIdx.x * blockDim.x;
	while (id < Amount_Of_Blocks)
	{
		array[id] = 10;
		id += blockDim.x * gridDim.x;
	}
}

void Foo_CPU(__int32* array)
{
	for (size_t i = 0; i < Amount_Of_Blocks; ++i)
	{
		_STD cout << array[i] << ' ' << i << NEW_LINE;
	}
}

////////////////////////////////////////////////////////
////////////////////////////////////////////////////////
/*
	PIXEL_GPU CLASS CASE
*/
////////////////////////////////////////////////////////
////////////////////////////////////////////////////////


__host__ __device__ Pixel_GPU::Pixel_GPU() :
	R(NULL),
	G(NULL),
	B(NULL),
	color_range(255)
{
	//Nothing here
}

__host__ __device__  Pixel_GPU::Pixel_GPU(const unsigned char R, const unsigned char G, const unsigned char B) :
	R(R),
	G(G),
	B(B),
	color_range(255)
{
	//Nothing here
}

__host__ __device__  Pixel_GPU::Pixel_GPU(const Pixel_GPU& Object) :
	R(Object.R),
	G(Object.G),
	B(Object.B),
	color_range(Object.color_range)
{
	//Nothing here
}

__host__ void Pixel_GPU::Show_Object() const
{
	_STD cout << "Red: [" << static_cast<int>(this->R) << " ] " << "Green: [" << static_cast<int>(this->G) << " ] " << "Blue: [" << static_cast<int>(this->B) << " ] " << "Color range: " << this->color_range << NEW_LINE;
}

__host__ __device__ void Pixel_GPU::Set_R(const unsigned char R)
{
	this->R = R;
}

__host__ __device__ void Pixel_GPU::Set_G(const unsigned char G)
{
	this->G = G;
}

__host__ __device__ void Pixel_GPU::Set_B(const unsigned char B)
{
	this->B = B;
}

__host__ __device__  void Pixel_GPU::Set_Color_Range(const int Color_Range)
{
	this->color_range = Color_Range;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_R() const
{
	return this->R;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_G() const
{
	return this->G;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_B() const
{
	return this->B;
}

__host__ __device__ const int Pixel_GPU::Get_Color_Range() const
{
	return this->color_range;
}

//__host__ __device__ Pixel_GPU * Pixel_GPU::Get_Pointer() const
//{
//	return ;
//}

__host__ __device__ Pixel_GPU& Pixel_GPU::operator=(const Pixel_GPU& Object)
{
	//if (this != _STD addressof(Object))
	if (this != &Object)
	{
		this->R = Object.R;
		this->G = Object.G;
		this->B = Object.B;
		this->color_range = Object.color_range;
	}
	return *this;
}

Pixel_GPU::~Pixel_GPU()
{
	this->R = NULL;
	this->B = NULL;
	this->B = NULL;
}


////////////////////////////////////////////////////////
////////////////////////////////////////////////////////






//RUBBISH
//char abc[4][4][4]{};// = { 1,2,3,4 };

	//int cos{97};
	//for (size_t i = 0; i < 4; i++)
	//{
	//	for (size_t j = 0; j < 4; j++)
	//	{
	//		for (size_t k = 0; k < 4; k++)
	//		{
	//			abc[i][j][k] = (char)(cos++);
	//		}
	//	
	//	}
	//}



	/*for (size_t i = 0; i < 4; i++)
	{
		for (size_t j = 0; j < 4; j++)
		{
			for (size_t k = 0; k < 4; k++)
			{
				_STD cout << abc[i][j][k] << ' ';
			}
			_STD cout << NEW_LINE;
		}
		_STD cout << NEW_LINE;
	}*/
	//_STD cout << abc[0][1][3] << ' ';
