#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#include "..//..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>



////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

class Pixel_GPU
{
private:
	/*
		PRIVATE VARIABLES
	*/
	unsigned char R;
	unsigned char G;
	unsigned char B;
	//////////////////////////////////////////////////////////////////////////////
protected:
	/*
		PROTECTED VARIABLES
	*/

	//////////////////////////////////////////////////////////////////////////////
public:
	/*
		KONSTRUKTORY
	*/
	__host__ __device__ Pixel_GPU();
	__host__ __device__ explicit Pixel_GPU(const unsigned char R, const unsigned char G, const unsigned char B);
	__host__ __device__ Pixel_GPU(const Pixel_GPU& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		FUNKCJE PUBLIC
	*/
	__host__ __device__ void Show_Object() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		SETTERY
	*/
	__host__ __device__ void Set_R(const unsigned char R);
	__host__ __device__ void Set_G(const unsigned char G);
	__host__ __device__ void Set_B(const unsigned char B);
	//////////////////////////////////////////////////////////////////////////////
	/*
		GETTERY
	*/
	__host__ __device__ const unsigned char Get_R() const;
	__host__ __device__ const unsigned char Get_G() const;
	__host__ __device__ const unsigned char Get_B() const;
	//__host__ __device__ Pixel_GPU * Get_Pointer() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		OPERATORY
	*/
	//JEDNOARGUMENTOWE
	__host__ __device__ Pixel_GPU& operator=(const Pixel_GPU& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		DESTRUKTOR
	*/
	virtual ~Pixel_GPU();
	//////////////////////////////////////////////////////////////////////////////
};

Pixel_GPU* Host_Array{};
size_t global_size{};

void Fill_Array(const _STD string& file_path);
__global__ void Counting_Unique_Colors(Pixel_GPU * Pixel_array, __int64 & unique_colors, const size_t & size);

int main(int argc, char* argv[])
{
	__int64 unique_colors{};
	__int64 unique{};
	size_t size{};

	HANDLE_ERROR(hipMalloc((void**)& unique_colors, sizeof(__int64)));
	HANDLE_ERROR(hipMalloc((void**)& size, sizeof(size_t)));

	unique_colors = 0;
	unique = 0;
	size = 0;

	Fill_Array("Lena.ppm");


	Pixel_GPU* Device_Array{};
	hipMalloc((void**)& Device_Array, global_size * sizeof(Pixel_GPU));


	hipMemcpy(Device_Array, Host_Array, global_size * sizeof(Pixel_GPU), HostToDevice);
	//hipMemcpy(&global_size, &size, sizeof(size_t), HostToDevice);
	//hipMemcpy(&size, &global_size, sizeof(size_t), HostToDevice);
	size = global_size;

	Counting_Unique_Colors <<<655,1024>>> (Device_Array, unique_colors, size);
	//Counting_Unique_Colors <<<1,1024>>> (Device_Array, unique_colors, size);



	//hipMemcpy(&unique, &unique_colors, sizeof(__int64), DeviceToHost);
	unique = unique_colors;
	_STD cout << unique << NEW_LINE;

	hipFree(&unique_colors);
	hipFree(&size);
	hipFree(Device_Array);

	free(Host_Array);

	system("pause");
	return 0;
}


void Fill_Array(const _STD string& file_path)
{
	_STD fstream file;
	file.open(file_path.c_str(), std::ios_base::in);
	_STD string line{};
	__int32 a{ 1 };
	Pixel_GPU Temporary_Pixel{};
	std::string extension{};					//extension (p1,p2,p3 etc)
	__int32 width{};							//width
	__int32 height{};							//height
	__int32 color_range{};						//color range
	static size_t i = 0;
	if (file.good() == false)
	{
		exit(0);
	}
	else
	{
		while (file.eof() == false)
		{
			if (a < 5)
			{
				_STD getline(file, line);
				if (a == 1)
				{
					extension = line;
				}
				else if (a == 3)
				{
					_STD string line_second = "";
					int counter = 0;
					for (int i = 0; i < line.size(); i++)
					{
						if (line[i] == ' ')
						{
							width = stoi(line_second);
							line_second = "";
						}
						else
						{
							line_second += line[i];
						}
						counter++;
					}
					height = stoi(line_second);
					global_size = static_cast<size_t>(width * height);
					Host_Array = (Pixel_GPU*)malloc(global_size * sizeof(Pixel_GPU));
				}
				else if (a == 4)
				{
					color_range = (stoi(line));
				}
			}
			else if (a >= 5)
			{
				int color = 0;
				file >> color;
				Temporary_Pixel.Set_R(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_G(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_B(color);
				color = 0;
				Host_Array[i++] = Temporary_Pixel;
			}
			line = "";
			a++;
		}
	}
	_STD cout << "--------------------------" << NEW_LINE;
	_STD cout << "  Information" << NEW_LINE;
	_STD cout << "  Loaded format: " << extension << NEW_LINE;
	_STD cout << "  Width: " << width << NEW_LINE;
	_STD cout << "  Height: " << height << NEW_LINE;
	_STD cout << "  Color Range: " << color_range << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	//_STD cout << "  Array size: " << static_cast<size_t>(Pixel_array.size()) << NEW_LINE;
	_STD cout << "  Array size: " << static_cast<size_t>((global_size)) << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	file.close();
}

__global__ void Counting_Unique_Colors(Pixel_GPU* Pixel_array, __int64 & unique_colors, const size_t& size)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	bool is_unique = true;
	//int j = id + 1;
	while (id < size)
	{
		is_unique = true;
		for (int j = id + 1; j < size; j += blockDim.x * gridDim.x)
		{
			if (Pixel_array[id].Get_R() == Pixel_array[j].Get_R() && Pixel_array[id].Get_G() == Pixel_array[j].Get_G() && Pixel_array[id].Get_B() == Pixel_array[j].Get_B())
			{
				is_unique = false;
				break;
			}
		}
		if (is_unique == true)
		{
			
		}
		//if (Pixel_array[id].Get_R() == Pixel_array[j].Get_R() && Pixel_array[id].Get_G() == Pixel_array[j].Get_G() && Pixel_array[id].Get_B() == Pixel_array[j].Get_B())
		//{
		//	is_unique = false;
		////	break;
		//}
		else
		{
			++unique_colors;
		}
		//unique_colors = id;
		id += blockDim.x * gridDim.x;
	}
}

////////////////////////////////////////////////////////
////////////////////////////////////////////////////////
/*
	PIXEL_GPU CLASS CASE
*/
////////////////////////////////////////////////////////
////////////////////////////////////////////////////////


__host__ __device__ Pixel_GPU::Pixel_GPU() :
	R(NULL),
	G(NULL),
	B(NULL)
{
	//Nothing here
}

__host__ __device__  Pixel_GPU::Pixel_GPU(const unsigned char R, const unsigned char G, const unsigned char B) :
	R(R),
	G(G),
	B(B)
{
	//Nothing here
}

__host__ __device__  Pixel_GPU::Pixel_GPU(const Pixel_GPU& Object) :
	R(Object.R),
	G(Object.G),
	B(Object.B)
{
	//Nothing here
}

__host__ __device__ void Pixel_GPU::Show_Object() const
{
	//_STD cout << "Red: [" << this->R << " ] " << "Green: [" << this->G << " ] " << "Blue: [" << this->B << " ] " << NEW_LINE;
}

__host__ __device__ void Pixel_GPU::Set_R(const unsigned char R)
{
	this->R = R;
}

__host__ __device__ void Pixel_GPU::Set_G(const unsigned char G)
{
	this->G = G;
}

__host__ __device__ void Pixel_GPU::Set_B(const unsigned char B)
{
	this->B = B;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_R() const
{
	return this->R;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_G() const
{
	return this->G;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_B() const
{
	return this->B;
}

//__host__ __device__ Pixel_GPU * Pixel_GPU::Get_Pointer() const
//{
//	return ;
//}

__host__ __device__ Pixel_GPU& Pixel_GPU::operator=(const Pixel_GPU& Object)
{
	//if (this != _STD addressof(Object))
	if (this != &Object)
	{
		this->R = Object.R;
		this->G = Object.G;
		this->B = Object.B;
	}
	return *this;
}

Pixel_GPU::~Pixel_GPU()
{
	this->R = NULL;
	this->B = NULL;
	this->B = NULL;
}


////////////////////////////////////////////////////////
////////////////////////////////////////////////////////