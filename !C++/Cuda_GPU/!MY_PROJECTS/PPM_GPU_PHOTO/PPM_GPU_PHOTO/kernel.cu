#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/device_functions.h"


#include "..//..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>



////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

class Pixel_GPU
{
private:
	/*
		PRIVATE VARIABLES
	*/
	unsigned char R;
	unsigned char G;
	unsigned char B;
	int color_range;
	//////////////////////////////////////////////////////////////////////////////
protected:
	/*
		PROTECTED VARIABLES
	*/

	//////////////////////////////////////////////////////////////////////////////
public:
	/*
		KONSTRUKTORY
	*/
	__host__ __device__ Pixel_GPU();
	__host__ __device__ explicit Pixel_GPU(const unsigned char R, const unsigned char G, const unsigned char B);
	__host__ __device__ Pixel_GPU(const Pixel_GPU& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		FUNKCJE PUBLIC
	*/
	__host__ void Show_Object() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		SETTERY
	*/
	__host__ __device__ void Set_R(const unsigned char R);
	__host__ __device__ void Set_G(const unsigned char G);
	__host__ __device__ void Set_B(const unsigned char B);
	__host__ __device__ void Set_Color_Range(const int Color_Range);
	//////////////////////////////////////////////////////////////////////////////
	/*
		GETTERY
	*/
	__host__ __device__ const unsigned char Get_R() const;
	__host__ __device__ const unsigned char Get_G() const;
	__host__ __device__ const unsigned char Get_B() const;
	__host__ __device__ const int Get_Color_Range() const;
	//__host__ __device__ Pixel_GPU * Get_Pointer() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		OPERATORY
	*/
	//JEDNOARGUMENTOWE
	__host__ __device__ Pixel_GPU& operator=(const Pixel_GPU& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		DESTRUKTOR
	*/
	virtual ~Pixel_GPU();
	//////////////////////////////////////////////////////////////////////////////
};

Pixel_GPU* Host_Array{};
size_t global_size{};

void Fill_Array(const _STD string& file_path);
void Show_Array(Pixel_GPU* Pixel_array, const size_t& size);
__global__ void Counting_Unique_Colors(Pixel_GPU * Pixel_array, __int64 * unique_colors, const size_t * size);

int main(int argc, char* argv[])
{
	//__int64 * unique_colors{};
	//__int64 unique{};
	//size_t * size{};

	//HANDLE_ERROR(hipMalloc((void**)& unique_colors, sizeof(__int64)));
	//HANDLE_ERROR(hipMalloc((void**)& size, sizeof(size_t)));

	////unique_colors = 0;
	////unique = 0;
	////size = 0;

	//Fill_Array("Lena.ppm");

	////Show_Array(Host_Array, global_size);

	//Pixel_GPU* Device_Array{};
	//hipMalloc((void**)& Device_Array, global_size * sizeof(Pixel_GPU));


	//hipMemcpy(Device_Array, Host_Array, global_size * sizeof(Pixel_GPU), HostToDevice);
	////hipMemcpy(&global_size, &size, sizeof(size_t), HostToDevice);
	//hipMemcpy(size, &global_size, sizeof(size_t), HostToDevice);
	////size = global_size;

	//Counting_Unique_Colors <<<655,1024>>> (Device_Array, unique_colors, size);
	////Counting_Unique_Colors <<<1,1024>>> (Device_Array, unique_colors, size);



	//hipMemcpy(Host_Array, Device_Array, global_size * sizeof(Pixel_GPU), DeviceToHost);
	//Show_Array(Host_Array, global_size);

	//hipMemcpy(&unique, unique_colors, sizeof(__int64), DeviceToHost);
	////unique = unique_colors;
	//_STD cout << unique << NEW_LINE;

	//hipFree(unique_colors);
	//hipFree(size);
	//hipFree(Device_Array);

	//free(Host_Array);

	char abc[4][4][4]{};// = { 1,2,3,4 };

	int cos{97};
	for (size_t i = 0; i < 4; i++)
	{
		for (size_t j = 0; j < 4; j++)
		{
			for (size_t k = 0; k < 4; k++)
			{
				abc[i][j][k] = (char)(cos++);
			}
		
		}
	}



	/*for (size_t i = 0; i < 4; i++)
	{
		for (size_t j = 0; j < 4; j++)
		{
			for (size_t k = 0; k < 4; k++)
			{
				_STD cout << abc[i][j][k] << ' ';
			}
			_STD cout << NEW_LINE;
		}
		_STD cout << NEW_LINE;
	}*/
	_STD cout << abc[0][1][3] << ' ';

	system("pause");
	return 0;
}


void Fill_Array(const _STD string& file_path)
{
	_STD fstream file;
	file.open(file_path.c_str(), std::ios_base::in);
	_STD string line{};
	__int32 a{ 1 };
	Pixel_GPU Temporary_Pixel{};
	std::string extension{};					//extension (p1,p2,p3 etc)
	__int32 width{};							//width
	__int32 height{};							//height
	__int32 color_range{};						//color range
	static size_t i = 0;
	if (file.good() == false)
	{
		exit(0);
	}
	else
	{
		while (file.eof() == false)
		{
			if (a < 5)
			{
				_STD getline(file, line);
				if (a == 1)
				{
					extension = line;
				}
				else if (a == 3)
				{
					_STD string line_second = "";
					int counter = 0;
					for (int i = 0; i < line.size(); i++)
					{
						if (line[i] == ' ')
						{
							width = stoi(line_second);
							line_second = "";
						}
						else
						{
							line_second += line[i];
						}
						counter++;
					}
					height = stoi(line_second);
					global_size = static_cast<size_t>(width * height);
					Host_Array = (Pixel_GPU*)malloc(global_size * sizeof(Pixel_GPU));
				}
				else if (a == 4)
				{
					color_range = (stoi(line));
				}
			}
			else if (a >= 5)
			{
				int color = 0;
				file >> color;
				Temporary_Pixel.Set_R(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_G(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_B(color);
				//Temporary_Pixel.Set_Color_Range(color_range);
				color = 0;
				Host_Array[i++] = Temporary_Pixel;
			}
			line = "";
			a++;
		}
	}
	_STD cout << "--------------------------" << NEW_LINE;
	_STD cout << "  Information" << NEW_LINE;
	_STD cout << "  Loaded format: " << extension << NEW_LINE;
	_STD cout << "  Width: " << width << NEW_LINE;
	_STD cout << "  Height: " << height << NEW_LINE;
	_STD cout << "  Color Range: " << color_range << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	//_STD cout << "  Array size: " << static_cast<size_t>(Pixel_array.size()) << NEW_LINE;
	_STD cout << "  Array size: " << static_cast<size_t>((global_size)) << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	file.close();
}

void Show_Array(Pixel_GPU* Pixel_array, const size_t& size)
{
	for (size_t i = 0; i < size; i++)
	{
		if (Pixel_array[i].Get_Color_Range() > 255)
		{
			Pixel_array[i].Show_Object();
			_STD cin.get();
		}
	/*	Pixel_array[i].Show_Object();
		_STD cin.get();*/
	}
}

__global__ void Counting_Unique_Colors(Pixel_GPU* Pixel_array, __int64 * unique_colors, const size_t * size)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	bool is_unique = true;
	const int color_range = Pixel_array[0].Get_Color_Range();
	if (id < *size)
	{
		//if (Pixel_array[id].Get_Color_Range() > color_range)
		//{
		//	continue;
		//}
		//else
		//{
		
		//}
		//__syncthreads();
		while (j < *size)
		{
			/*	Pixel_array[j].Set_Color_Range(static_cast<int>(color_range + 100));
				Pixel_array[j].Set_R(static_cast<int>(color_range + 100));*/
		/*	if (Pixel_array[id].Get_R() == Pixel_array[j].Get_R() && Pixel_array[id].Get_G() == Pixel_array[j].Get_G() && Pixel_array[id].Get_B() == Pixel_array[j].Get_B())
			{
				Pixel_array[j].Set_Color_Range(static_cast<int>(color_range + 100));
			}*/
			Pixel_array[j].Set_Color_Range(static_cast<int>(color_range + 100));
			j++;
		}
		//__syncthreads();
		//Pixel_array[id].Set_Color_Range(static_cast<int>(color_range + 100));
		//__syncthreads();
		/*Pixel_array[id].Set_Color_Range(static_cast<int>(color_range + 100));
		Pixel_array[id].Set_R(static_cast<int>(color_range + 100));
		id += blockDim.x * gridDim.x;*/
	}

	id = threadIdx.x + blockIdx.x * blockDim.x;
	if (Pixel_array[id].Get_Color_Range() == color_range)
	{
		++(*unique_colors);
	}
	//(*unique_colors) = 1000;

}

////////////////////////////////////////////////////////
////////////////////////////////////////////////////////
/*
	PIXEL_GPU CLASS CASE
*/
////////////////////////////////////////////////////////
////////////////////////////////////////////////////////


__host__ __device__ Pixel_GPU::Pixel_GPU() :
	R(NULL),
	G(NULL),
	B(NULL),
	color_range(255)
{
	//Nothing here
}

__host__ __device__  Pixel_GPU::Pixel_GPU(const unsigned char R, const unsigned char G, const unsigned char B) :
	R(R),
	G(G),
	B(B),
	color_range(255)
{
	//Nothing here
}

__host__ __device__  Pixel_GPU::Pixel_GPU(const Pixel_GPU& Object) :
	R(Object.R),
	G(Object.G),
	B(Object.B),
	color_range(Object.color_range)
{
	//Nothing here
}

__host__ void Pixel_GPU::Show_Object() const
{
	_STD cout << "Red: [" << static_cast<int>(this->R) << " ] " << "Green: [" << static_cast<int>(this->G) << " ] " << "Blue: [" << static_cast<int>(this->B) << " ] " << "Color range: " << this->color_range << NEW_LINE;
}

__host__ __device__ void Pixel_GPU::Set_R(const unsigned char R)
{
	this->R = R;
}

__host__ __device__ void Pixel_GPU::Set_G(const unsigned char G)
{
	this->G = G;
}

__host__ __device__ void Pixel_GPU::Set_B(const unsigned char B)
{
	this->B = B;
}

__host__ __device__  void Pixel_GPU::Set_Color_Range(const int Color_Range)
{
	this->color_range = Color_Range;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_R() const
{
	return this->R;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_G() const
{
	return this->G;
}

__host__ __device__ const unsigned char Pixel_GPU::Get_B() const
{
	return this->B;
}

__host__ __device__ const int Pixel_GPU::Get_Color_Range() const
{
	return this->color_range;
}

//__host__ __device__ Pixel_GPU * Pixel_GPU::Get_Pointer() const
//{
//	return ;
//}

__host__ __device__ Pixel_GPU& Pixel_GPU::operator=(const Pixel_GPU& Object)
{
	//if (this != _STD addressof(Object))
	if (this != &Object)
	{
		this->R = Object.R;
		this->G = Object.G;
		this->B = Object.B;
		this->color_range = Object.color_range;
	}
	return *this;
}

Pixel_GPU::~Pixel_GPU()
{
	this->R = NULL;
	this->B = NULL;
	this->B = NULL;
}


////////////////////////////////////////////////////////
////////////////////////////////////////////////////////