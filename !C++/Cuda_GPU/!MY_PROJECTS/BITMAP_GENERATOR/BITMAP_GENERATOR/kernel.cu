
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/device_functions.h"


#include "..//..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>


////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define N 100
#define Amount_Of_Blocks 655
#define Amount_Of_Threads 1024

//SELF HEADERS
#include "Functions.hpp"
////////////////////////////////////

int main(int argc, char* argv[])
{
	using Functions::Read_BitMap;

	_STD string file_path = "bitmap1.png";

	Read_BitMap(file_path.c_str());

	system("pause");
	return 0;
}