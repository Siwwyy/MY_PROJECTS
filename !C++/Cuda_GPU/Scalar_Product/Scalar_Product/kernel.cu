#include "hip/hip_runtime.h"
﻿//#define __cplusplus
//#define __HIPCC__
//#define __CUDACC_RTC__

#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define imin(a,b) (a<b?a:b)
#define sum_squares(x)  (x*(x+1)*(2*x+1)/6)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);


__global__ void dot(float* a, float* b, float* c) 
{
	__shared__ float cache[threadsPerBlock]{};
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp{};
	while (tid < N) 
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	// Ustawienie wartoœci pamiêci podrêcznej
	cache[cacheIndex] = temp;

	// Synchronizacja w¹tków w tym bloku
	__syncthreads();

	// W przypadku redukcji threadsPerBlock musi byæ potêg¹ 2,
	// ze wzglêdu na poni¿szy kod
	int i = blockDim.x / 2;
	while (i != 0) 
	{
		if (cacheIndex < i)
		{
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
	{
		c[blockIdx.x] = cache[0];
	}		
}

int main(int argc, char* argv[])
{
	float* a{}, * b{}, c{}, * partial_c{};
	float* dev_a{}, * dev_b{}, * dev_partial_c{};

	// Alokacja pamiêci na CPU
	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

	// Alokacja pamiêci na GPU
	HANDLE_ERROR(hipMalloc((void**)& dev_a, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)& dev_b, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)& dev_partial_c, blocksPerGrid * sizeof(float)));

	// Zape³nienie pamiêci hosta danymi
	for (int i = 0; i < N; i++) 
	{
		a[i] = i;
		b[i] = i * 2;
	}

	// Skopiowanie tablic a i b do GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

	dot<<<blocksPerGrid, threadsPerBlock>>> (dev_a, dev_b,dev_partial_c);

	// Skopiowanie tablicy c z powrotem z GPU do CPU
	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

	// Zakoñczenie pracy na CPU
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++) 
	{
		c += partial_c[i];
	}


	printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares((float)(N - 1)));

	// Zwolnienie pamiêci na GPU
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_partial_c));

	// Zwolnienie pamiêci na CPU
	free(a);
	free(b);
	free(partial_c);

	system("pause");
	return 0;
}