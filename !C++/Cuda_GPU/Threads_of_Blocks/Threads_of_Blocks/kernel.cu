#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
//#define N 10
#define N (33 * 1024)

__global__ void addition(__int32* array_1, __int32* array_2, __int32* array_3);

int main(int argc, char* argv[])
{
	/*_STD cout << argc << NEW_LINE;
	_STD cout << *argv << NEW_LINE;*/
	int a[N]{}, b[N]{}, c[N]{};
	int* dev_a{}, * dev_b{}, * dev_c{};
	// Alokacja pamięci na GPU
	HANDLE_ERROR(hipMalloc((void**)& dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)& dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)& dev_c, N * sizeof(int)));
	//Zapełnienie tablic a i b na CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i + i;
	}
	//Kopiowanie tablic a i b do GPU
	//_STD cout << ((N + (N - 1)) / 2) << NEW_LINE;
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
	//addition <<<((N + (N-1))/2),N>>> (dev_a, dev_b, dev_c);
	addition <<<128,128>>> (dev_a, dev_b, dev_c);
	// Kopiowanie tablicy c z GPU do CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
	// Wyświetlenie wyniku
	for (int i = 0; i < N; i++)
	{
		printf_s("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	// Zwolnienie pamięci alokowanej na GPU
	//std::cout << hipDeviceAttributeMaxThreadsPerBlock << NEW_LINE;
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	////Some informations about GPU capacity (max threads etc.)
	//_STD cout << NEW_LINE;
	//dim3 gridDim{N,N};
	//dim3 blockDim{N,N};
	//printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
	//hipDeviceProp_t prop{};
	//int count{};
	//HANDLE_ERROR(hipGetDeviceCount(&count));	//amount of Device properties
	//for (int i = 0; i < count; i++)
	//{
	//	HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
	//	printf("Amount of global memory: %ld\n", prop.totalGlobalMem);
	//	printf("Amount of constant memory: %ld\n", prop.totalConstMem);
	//	printf("Max. memory height: %ld\n", prop.memPitch);
	//	printf("Max. amount of threads per block %d\n", prop.maxThreadsPerBlock);
	//	printf("Max. amount of threads dimensionals: (%d, %d, %d)\n",prop.maxThreadsDim[0], prop.maxThreadsDim[i],prop.maxThreadsDim[2]);
	//	printf("Max. amount of grid dimensionals: (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[i], prop.maxGridSize[2]);
	//	printf("\n");
	//}

	system("pause");
	return 0;
}

//__global__ void addition(__int32* array_1, __int32* array_2, __int32* array_3)
//{
//	int id = threadIdx.x;
//	if (id < N)
//	{
//		//add *100000 or more to get an error execution
//		array_3[id] = (array_1[id] + array_2[id]);
//	}
//	/*
//		It means that when we are invoking this function (less than N - 1 off course,
//		cause our array size is N) we have to call blocks of GPU's contained in GRID.
//		int id = threadIdx.x is just a number of thread in currently currently 
//		(in easy way, index) weworking on
//	*/
//}

__global__ void addition(__int32* array_1, __int32* array_2, __int32* array_3)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	while(id < N) 
	{
		array_3[id] = array_1[id] + array_2[id];
		id += blockDim.x * gridDim.x;
		//id = threadIdx.x + blockIdx.x * blockDim.x;
	}
	/*
		It means that when we are invoking this function (less than N - 1 off course,
		cause our array size is N) we have to call blocks of GPU's contained in GRID.
		int id = threadIdx.x is just a number of thread in currently currently
		(in easy way, index) weworking on
	*/
}

