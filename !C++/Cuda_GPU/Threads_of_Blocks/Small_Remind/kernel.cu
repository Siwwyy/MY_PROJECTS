#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <string>
#include <iostream>


////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define N 10	//size of array

__global__ void Multiply(__int64* array_1, __int64* array_2, __int64* array_3);

int main(int argc, char* argv[])
{
	//__int64 * a_CPU{}, * b_CPU{}, * c_CPU{};
	__int64 a_CPU[N][N], b_CPU[N][N], c_CPU[N*N];
	__int64 * a_GPU{}, * b_GPU{}, * c_GPU{};

	/*a_CPU = (__int64*)malloc(N * sizeof(__int64));
	b_CPU = (__int64*)malloc(N * sizeof(__int64));
	c_CPU = (__int64*)malloc(N * sizeof(__int64));*/

	hipMalloc((void**)& a_GPU, N * N * sizeof(__int64));
	hipMalloc((void**)& b_GPU, N * N * sizeof(__int64));
	hipMalloc((void**)& c_GPU, N * N * sizeof(__int64));

	//for (size_t i = 0; i < N; ++i)
	//{
	//	*(a_CPU + i) = static_cast<__int64>(i);
	//	*(b_CPU + i) = static_cast<__int64>(i * i);
	///*	a_CPU[i] = static_cast<__int64>(i);
	//	b_CPU[i] = static_cast<__int64>(i*i);*/
	//	*(c_CPU + i) = 0;
	//}

	__int64 value = 0;
	for (size_t i = 0; i < N; ++i)
	{
		for (size_t j = 0; j < N; ++j)
		{
			//a_CPU[i][j] = static_cast<__int64>(j+i);
			a_CPU[i][j] = static_cast<__int64>(1);
			//b_CPU[i][j] = static_cast<__int64>(i*j);
			b_CPU[i][j] = static_cast<__int64>(2);
			//*(*(c_CPU + i) + j) = 0;
			*(c_CPU + i+j) = 0;
			value++;
		}
	}

	hipMemcpy(a_GPU, a_CPU, N * N * sizeof(__int64), HostToDevice);
	hipMemcpy(b_GPU, b_CPU, N * N * sizeof(__int64), HostToDevice);

	//memcpy(c_CPU, a_CPU, N * N * sizeof(__int64));

	Multiply <<<N, N>>> (a_GPU, b_GPU, c_GPU);

	hipMemcpy(c_CPU, c_GPU, N * N * sizeof(__int64), DeviceToHost);

	//for (size_t i = 0; i < N; ++i)
	//{
	//	for (size_t j = 0; j < N; ++j)
	//	{
	//	
	//		_STD cout << *(*(a_CPU + i) + j) << ' ';
	//	}
	//	_STD cout << NEW_LINE;
	//}

	for (size_t i = 0; i < N*N; ++i)
	{
		if (i % (N) == 0 && i > 1)
		{
			_STD cout << NEW_LINE;
		}
		_STD cout << *(c_CPU+i) << ' ';
		//for (size_t j = 0; j < N; ++j)
		//{
		////	_STD cout << *(*(c_CPU+i) + j) << ' ';
		//}
		////_STD cout << NEW_LINE;
	
	}
	_STD cout << NEW_LINE;
	//free(a_CPU);
	//free(b_CPU);
	//free(c_CPU);

	hipFree(a_GPU);
	hipFree(b_GPU);
	hipFree(c_GPU);

	system("pause");    
	return 0;
}

__global__ void Multiply(__int64* array_1, __int64* array_2, __int64* array_3)
{
	size_t index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < N+index)
	{
		array_3[index] = array_1[index] + array_2[index];
	}

	/*size_t index_x = blockIdx.x;
	size_t index_y = threadIdx.x + blockIdx.x * blockDim.x;
	while (index_y < N)
	{
		array_3[index_x][index_y] = array_1[index_x][index_y] + array_2[index_x][index_y];
		index_y += blockDim.x * gridDim.x;
	}*/
}