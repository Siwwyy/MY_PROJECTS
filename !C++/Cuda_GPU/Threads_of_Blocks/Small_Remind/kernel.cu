#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <string>
#include <iostream>


////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define N 10	//size of array

__global__ void Multiply(__int64* array_1, __int64* array_2, __int64* array_3);

int main(int argc, char* argv[])
{
	//__int64 * a_CPU{}, * b_CPU{}, * c_CPU{};
	__int64 a_CPU[N][N], b_CPU[N][N], c_CPU[N][N];
	__int64 * a_GPU{}, * b_GPU{}, * c_GPU{};

	/*a_CPU = (__int64*)malloc(N * sizeof(__int64));
	b_CPU = (__int64*)malloc(N * sizeof(__int64));
	c_CPU = (__int64*)malloc(N * sizeof(__int64));*/

	hipMalloc((void**)& a_GPU, N*N * sizeof(__int64));
	hipMalloc((void**)& b_GPU, N * N * sizeof(__int64));
	hipMalloc((void**)& c_GPU, N * N * sizeof(__int64));

	//for (size_t i = 0; i < N; ++i)
	//{
	//	*(a_CPU + i) = static_cast<__int64>(i);
	//	*(b_CPU + i) = static_cast<__int64>(i * i);
	///*	a_CPU[i] = static_cast<__int64>(i);
	//	b_CPU[i] = static_cast<__int64>(i*i);*/
	//	*(c_CPU + i) = 0;
	//}

	for (size_t i = 0; i < N; ++i)
	{
		for (size_t j = 0; j < N; ++j)
		{
			a_CPU[i][j] = static_cast<__int64>(j+i);
			b_CPU[i][j] = static_cast<__int64>(i*j);
			*(*(c_CPU + i) + j) = 0;

		}
	}

	hipMemcpy(a_GPU, a_CPU, N * N * sizeof(__int64), HostToDevice);
	hipMemcpy(b_GPU, b_CPU, N * N * sizeof(__int64), HostToDevice);

	Multiply <<<N, N>>> (a_GPU, b_GPU, c_GPU);

	hipMemcpy(c_CPU, c_GPU, N * N * sizeof(__int64), DeviceToHost);

	for (size_t i = 0; i < N; ++i)
	{
		for (size_t j = 0; j < N; ++j)
		{
			_STD cout << *(*(c_CPU+i) + j) << ' ';
		}
		_STD cout << NEW_LINE;
	}

	//free(a_CPU);
	//free(b_CPU);
	//free(c_CPU);

	hipFree(a_GPU);
	hipFree(b_GPU);
	hipFree(c_GPU);

	system("pause");    
	return 0;
}

__global__ void Multiply(__int64* array_1, __int64* array_2, __int64* array_3)
{
	size_t index = threadIdx.x + blockIdx.x * blockDim.x;
	while (index < N)
	{
		array_3[index] = array_1[index] + array_2[index];
		index += blockDim.x * gridDim.x;
	}


	/*size_t index_x = blockIdx.x;
	size_t index_y = threadIdx.x + blockIdx.x * blockDim.x;
	while (index_y < N)
	{
		array_3[index_x][index_y] = array_1[index_x][index_y] + array_2[index_x][index_y];
		index_y += blockDim.x * gridDim.x;
	}*/
}