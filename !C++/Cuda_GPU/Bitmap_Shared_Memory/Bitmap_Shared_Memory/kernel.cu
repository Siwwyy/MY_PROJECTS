#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/hip_runtime_api.h"


#include "..//..//common/book.h"
#include "..//..//common/cpu_bitmap.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>



////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define DIM 1024
#define PI 3.1415926535897932f

//
//__global__ void kernel(unsigned char* ptr) 
//{
//	// Rzutowanie threadIdx/BlockIdx na pozycje pikseli
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int offset = x + y * blockDim.x * gridDim.x;
//
//	__shared__ float shared[16][16];
//
//	// Obliczenie warto�ci dla tej pozycji
//	const float period = 128.0f;
//
//	shared[threadIdx.x][threadIdx.y] = 255 * (sinf(x * 2.0f * PI / period) + 1.0f) * (sinf(y * 2.0f * PI / period) + 1.0f) / 4.0f;
//
//	// Usuni�cie tego wywo�ania funkcji syncthreads b�dzie widoczne
//	// w wyniku. Dlatego jest ona potrzebna.
//	__syncthreads();
//
//	ptr[offset * 4 + 0] = 0;
//	ptr[offset * 4 + 1] = shared[15 - threadIdx.x][15 - threadIdx.y];
//	ptr[offset * 4 + 2] = 0;
//	ptr[offset * 4 + 3] = 255;
//}
//
//// Dane globalne potrzebne procedurze aktualizuj�cej
//struct DataBlock 
//{
//	unsigned char* dev_bitmap;
//};
//
//int main(int argc, char* argv[])
//{
//	DataBlock   data;
//	CPUBitmap bitmap(DIM, DIM, &data);
//	unsigned char* dev_bitmap;
//
//	HANDLE_ERROR(hipMalloc((void**)& dev_bitmap, bitmap.image_size())); 
//	data.dev_bitmap = dev_bitmap;
//
//	dim3 grids(DIM / 16, DIM / 16);
//	dim3 threads(16, 16);
//	kernel <<<grids, threads >>> (dev_bitmap);
//
//	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));
//
//	HANDLE_ERROR(hipFree(dev_bitmap));
//
//	bitmap.display_and_exit();
//
//	system("pause");
//	return 0;
//}


#define DIM 1000

struct hipComplex {
    float   r;
    float   i;
    // hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ hipComplex(float a, float b) : r(a), i(b) {} // Fix error for calling host function from device
    __device__ float magnitude2(void) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i = 0; i < 200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel(unsigned char* ptr) {
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia(x, y);
    ptr[offset * 4 + 0] = 255 * juliaValue;
    ptr[offset * 4 + 1] = 0;
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char* dev_bitmap;
};

int main(void) {
    DataBlock   data;
    CPUBitmap bitmap(DIM, DIM, &data);
    unsigned char* dev_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));
    data.dev_bitmap = dev_bitmap;

    dim3    grid(DIM, DIM);
    kernel << <grid, 1 >> > (dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
        bitmap.image_size(),
        hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_bitmap));

    bitmap.display_and_exit();
}
