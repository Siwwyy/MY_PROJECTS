#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"
#include "..//..//common/cpu_anim.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define DIM 1920

struct DataBlock
{
	unsigned char* dev_bitmap;
	CPUAnimBitmap* bitmap;

};

// Zwolnienie pamięci alokowanej na GPU
void cleanup(DataBlock* d) 
{
	hipFree(d->dev_bitmap);
}


__global__ void kernel(unsigned char* ptr, int ticks)
{
	// Odwzorowanie z threadldx/Blockldx na położenie pikseli
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	// Obliczenie wartości dla danego miejsca
	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));
	ptr[offset * 4 + 0] = grey;
	ptr[offset * 4 + 1] = grey;
	ptr[offset * 4 + 2] = grey;
	ptr[offset * 4 + 3] = 255;
}


void generate_frame(DataBlock* d, int ticks) {
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel<<<blocks, threads>>>(d->dev_bitmap, ticks);
	HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(),d->dev_bitmap,d->bitmap->image_size(),hipMemcpyDeviceToHost));
}


int main(int argc, char* argv[])
{
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	HANDLE_ERROR(hipMalloc((void**)& data.dev_bitmap, bitmap.image_size()));
	bitmap.anim_and_exit((void (*)(void*, int))generate_frame,(void (*)(void*))cleanup);

	system("pause");
	return 0;
}