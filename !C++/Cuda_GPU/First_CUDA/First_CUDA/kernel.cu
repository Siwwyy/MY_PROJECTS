#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <windows.h>

#include "..//..//common/book.h"

__global__ void Hello_WORD_GPU(void)
//__global__ void Hello_WORD_GPU(int & i)
{
	//static int i;
	printf("HELLO WORD GPU ! \n");
	//printf("HELLO WORD GPU !%\d \n", i++);
}

//void Hello_WORD_CPU()
//{
//	printf("HELLO WORD CPU ! \n");
//}

//int main()
//{
//	//Hello_WORD_CPU();
////	int i = 0;
//	printf("HELLO WORD CPU ! \n");
//	Hello_WORD_GPU <<<10, 1>>>();;
//	hipDeviceReset();
//	system("pause");
//    return 0;
//}

//int main()
//{
//	//Hello_WORD_CPU();
////	int i = 0;
//	hipDeviceProp_t prop{};
//	int count{};
//	HANDLE_ERROR(hipGetDeviceCount(&count));
//	for (int i = 0; i < count; i++)
//	{
//		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
//		//Kod wykorzystujący zdobyte informacje o właściwościach
//	}
//	system("pause");
//	return 0;
//}

int main()
{
	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++)
	{
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf(" --- Ogólne informacje o urządzeniu %d — \n", i);
		printf("Nazwa: %s\n", prop.name);
		printf("Potencjal obliczeniowy: %d.%d\n", prop.major, prop.minor);
		printf("Zegar: %d\n", prop.clockRate);
		printf("Ustawienie deviceOverlap: ");
		if (prop.deviceOverlap)
			printf("Wlączone\n");
		else
			printf("Wylączone\n");

		printf("Limit czasu dziatania jądra: ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Wylaczony\n");
		else
			printf("Wylaczony\n");
		printf("Informacje o pamięci urządzenia %d '\n", i);
		printf("Ilość pamięci globalnej: %ld\n", prop.totalGlobalMem);
		printf("Ilość pamięci stalej: %ld\n", prop.totalConstMem);
		printf("Maks. szerokość pamięci: %ld\n", prop.memPitch);
		printf("Wyrównanie tekstur: %ld\n", prop.textureAlignment);
		printf("Informacje na temat wieloprocesorów urządzenia %d \n", i);
		printf("Liczba wieloprocesorów: %d\n",
			prop.multiProcessorCount);
		printf("Pamięć wspolna na wieloprocesor: %ld\n", prop.sharedMemPerBlock);
		printf("Rejestry na wieloprocesor: %d\n", prop.regsPerBlock);
		printf("Liczba wątków w osnowie: %d\n", prop.warpSize);
		printf("Maks. liczba watkow na blok: %d\n",
			prop.maxThreadsPerBlock);
		printf("Maks. liczba wymiarów wątków: (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[i],
			prop.maxThreadsDim[2]);
		printf("Maks. liczba wymiarów siatki: (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[i],
			prop.maxGridSize[2]);
		printf("\n");
	}
	system("pause");
	return 0;
}