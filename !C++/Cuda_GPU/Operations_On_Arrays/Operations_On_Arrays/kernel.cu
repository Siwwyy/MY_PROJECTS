#include "hip/hip_runtime.h"
//#define __cplusplus
//#define __HIPCC__
//#define __CUDACC_RTC__

#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"

#include "..//..//Operations_On_Arrays/Operations_On_Arrays/Pixel.hpp"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <string>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'



void Fill_Array(const _STD string& file_path, _STD vector<RGB::Pixel> & Pixel_array);
__global__ void Counting_Unique_Colors(_STD vector<RGB::Pixel> & Pixel_array);

int main(int argc, char* argv[])
{
	_STD vector<RGB::Pixel> Pixel_array{};		//vector klasy Pixel

	Fill_Array("Lena.ppm", Pixel_array);
	system("pause");
	return 0;
}

void Fill_Array(const _STD string & file_path, _STD vector<RGB::Pixel>& Pixel_array)
{
	_STD fstream file;
	file.open(file_path.c_str(), std::ios_base::in);
	_STD string line{};
	__int32 a{ 1 };
	__int32 licznik{};
	__int32 char_number{};
	RGB::Pixel Temporary_Pixel{};
	std::string extension{};					//extension (p1,p2,p3 etc)
	__int32 width{};							//width
	__int32 height{};							//height
	__int32 color_range{};						//color range

	if (file.good() == false)
	{
		exit(0);
	}
	else
	{
		while (file.eof() == false)
		{
			if (a < 5)
			{
				_STD getline(file, line);
				if (a == 1)
				{
					extension = line;
				}
				else if (a == 3)
				{
					_STD string line_second = "";
					int counter = 0;
					for (int i = 0; i < line.size(); i++)
					{
						if (line[i] == ' ')
						{
							width = stoi(line_second);
							line_second = "";
						}
						else
						{
							line_second += line[i];
						}
						counter++;
					}
					height = stoi(line_second);
					Pixel_array.reserve(width * height);
				}
				else if (a == 4)
				{
					color_range = (stoi(line));
				}
			}
			else if (a >= 5)
			{
				int color = 0;
				file >> color;
				Temporary_Pixel.Set_R(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_G(color);
				color = 0;
				file >> color;
				Temporary_Pixel.Set_B(color);
				color = 0;
				Pixel_array.emplace_back(Temporary_Pixel);
			}
			line = "";
			a++;
		}
	}
	_STD cout << "--------------------------" << NEW_LINE;
	_STD cout << "  Information" << NEW_LINE;
	_STD cout << "  Loaded format: " << extension << NEW_LINE;
	_STD cout << "  Width: " << width << NEW_LINE;
	_STD cout << "  Height: " << height << NEW_LINE;
	_STD cout << "  Color Range: " << color_range << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	_STD cout << "  Array size: " << static_cast<size_t>(Pixel_array.size()) << NEW_LINE;
	_STD cout << "------------------------" << NEW_LINE;
	file.close();
}

__global__ void Counting_Unique_Colors(::std::vector<RGB::Pixel>& Pixel_array)
{
	
}