#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/device_functions.h"


#include "..//..//common/book.h"
#include "..//..//common/cpu_bitmap.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>




////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define DIM 1024

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

struct Sphere {
	float   r, b, g;
	float   radius;
	float   x, y, z;
	__device__ float hit(float ox, float oy, float* n) {
		float dx = ox - x;
		float dy = oy - y;
		if (dx * dx + dy * dy < radius * radius) {
			float dz = sqrtf(radius * radius - dx * dx - dy * dy);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}
		return -INF;
	}
};
#define SPHERES 20

__constant__ Sphere s[SPHERES];

__global__ void kernel(unsigned char* ptr) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float   ox = (x - DIM / 2);
	float   oy = (y - DIM / 2);

	float   r = 0, g = 0, b = 0;
	float   maxz = -INF;
	for (int i = 0; i < SPHERES; i++) {
		float   n;
		float   t = s[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	}

	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;
}

// Dane globalne potrzebne procedurze aktualizuj�cej
struct DataBlock {
	unsigned char* dev_bitmap;
};

int main(int argc, char* argv[])
{
	
	DataBlock   data;
	// Zarejestrowanie czasu pocz�tkowego
	hipEvent_t     start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	CPUBitmap bitmap(DIM, DIM, &data);
	unsigned char* dev_bitmap;

	// Alokowanie pami�ci na GPU dla mapy bitowej
	HANDLE_ERROR(hipMalloc((void**)& dev_bitmap,
		bitmap.image_size()));

	// Alokowanie tymczasowej pami�ci, zainicjowanie jej, skopiowanie jej do
	// pami�ci sta�ej na GPU, a nast�pnie zwolnienie
	Sphere* temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
	for (int i = 0; i < SPHERES; i++) {
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(1000.0f) - 500;
		temp_s[i].y = rnd(1000.0f) - 500;
		temp_s[i].z = rnd(1000.0f) - 500;
		temp_s[i].radius = rnd(100.0f) + 20;
	}
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sizeof(Sphere) * SPHERES));
	free(temp_s);

	// Wygenerowanie mapy bitowej z danych kul
	dim3    grids(DIM / 16, DIM / 16);
	dim3    threads(16, 16);
	kernel << <grids, threads >> > (dev_bitmap);

	// Skopiowanie mapy bitowej z powrotem z GPU w celu wy�wietlenia
	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
		bitmap.image_size(),
		hipMemcpyDeviceToHost));

	// Zarejestrowanie czasu zako�czenia i wy�wietlenie wyniku pomiaru czasu
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
		start, stop));
	printf("Czas generowania:  %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	HANDLE_ERROR(hipFree(dev_bitmap));

	// Wy�wietlenie
	bitmap.display_and_exit();
	system("pause");
	return 0;
}