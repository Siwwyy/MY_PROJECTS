#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

using namespace std;

////////////////////////////////////////////////////
//GPU

#define NEW_LINE '\n'


#define D2H hipMemcpyDeviceToHost
#define H2D hipMemcpyHostToDevice

#define BLOCK_SIZE 10	//height
#define GRID_SIZE 10	//width

//__global__ void Fill_Array(int** array_2D);
//__global__ void Multiply(int** array_2D, int n);

int main(int argc, char* argv[])
{
	//int **array_2D_GPU;
	//int array_2D_CPU[GRID_SIZE][BLOCK_SIZE]{};
	////constexpr size_t array_2D_GPU_size_i = DIM;

	//dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // so your threads are BLOCK_SIZE*BLOCK_SIZE, 256 in this case
	//dim3 dimGrid(GRID_SIZE, GRID_SIZE); // 1*1 blocks in a grid

	//HANDLE_ERROR(hipMalloc((void***)& array_2D_GPU, array_2D_GPU_size));
	////HANDLE_ERROR(hipMalloc((void**)&array_2D, array_2D_size * sizeof(int)));
	////HANDLE_ERROR(hipMalloc((void**)&array_2D, DIM * sizeof(int)));

	////Fill the array
	///*Fill_Array<<<DIM, DIM>>>(array_2D_GPU);
	//Multiply<<<DIM, DIM>>>(array_2D_GPU,2);*/

	//HANDLE_ERROR(hipMemcpy(array_2D_CPU, array_2D_GPU, array_2D_GPU_size, hipMemcpyDeviceToHost));
	//// Wyświetlenie wyniku
	//for (size_t i = 0; i < array_2D_GPU_size; i++)
	//{
	//	std::cout << '|';
	//	for (size_t j = 0; j < array_2D_GPU_size; j++)
	//	{
	//		std::cout << array_2D_CPU[i][j] << '|';
	//	}
	//	std::cout << NEW_LINE;
	//}

	//hipFree(array_2D_GPU);
	//FOR GPU
	//int** array_2D_GPU;
	//int** array_2D_CPU;
	//constexpr size_t array_size_GPU = (GRID_SIZE * BLOCK_SIZE * sizeof(int));

	////allocation
	//HANDLE_ERROR(hipMalloc((void**)&array_2D_GPU, (GRID_SIZE*sizeof(int*))));
	//HANDLE_ERROR(hipMemcpy(array_2D_GPU, array_2D_CPU, array_size_GPU, D2H));
	//for (size_t i = 0; i < GRID_SIZE; i++)
	//{
	//	hipMalloc((void**)&array_2D_GPU[i], BLOCK_SIZE * sizeof(int));
	//}
	//HANDLE_ERROR(hipMemcpy(array_2D_CPU, array_2D_GPU, array_size_GPU, H2D));
	int** ppArray_a{}, ** ppArray_b{};
	const size_t size = 10;
	hipMalloc((void**)& ppArray_a, 10 * sizeof(int*));

	for (int i = 0; i < 10; i++)

	{

		hipMalloc(&ppArray_b[i], 100 * sizeof(int)); /* Replace 100 with the dimension that u want */

	}

	hipMemcpy(ppArray_a, ppArray_b, 10 * sizeof(int*), H2D);
	//FOR CPU
	//int array_2D_CPU[GRID_SIZE][BLOCK_SIZE]{};
	//constexpr size_t array_size_CPU = (GRID_SIZE * BLOCK_SIZE * sizeof(int));
	//int* dev_a{};
	//// Alokacja pamięci na GPU
	//HANDLE_ERROR(hipMalloc((void**)& dev_a, 10 * sizeof(int)));
	//hipFree(array_2D_GPU);
	system("pause");
	return 0;
}

/////////////////////////////////////////////////////////////////