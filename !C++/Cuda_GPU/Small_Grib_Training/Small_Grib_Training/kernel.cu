#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "..//..//common/book.h"

#include <stdio.h>
#include <windows.h>
#include <iostream>

constexpr size_t array_size = 10;
#define N array_size
///*
//	ON CPU
//*/
//
//void addition(__int32 * array_1, __int32 * array_2, __int32 * array_3);
//
//int main(int argc, char * argv[])
//{
//	__int32 array_1[array_size]{}, array_2[array_size]{}, array_3[array_size]{};
//		
//	for (size_t i = 0; i < array_size; ++i)
//	{
//		array_1[i] = i;
//		array_2[i] = i+i;
//	}
//
//	addition(array_1, array_2, array_3);
//	//Dispalying the results of adding 
//	for (size_t i = 0; i < array_size; ++i)
//	{
//		printf_s("%d + %d = %d\n", array_1[i], array_2[i], array_3[i]);
//	}
//
//	system("pause");
//	return 0;
//}
//
//void addition(__int32* array_1, __int32* array_2, __int32* array_3)
//{
//	/*for (size_t i = 0; i < array_size; ++i)
//	{
//		array_3[i] = (array_1[i] + array_2[i]);
//	}*/
//	size_t id{};
//	while(id < array_size)
//	{
//		array_3[id] = (array_1[id] + array_2[id]);
//		++id;
//	}
//}

__global__ void addition(__int32* array_1, __int32* array_2, __int32* array_3);

int main(int argc, char * argv[])
{
	int a[N]{}, b[N]{}, c[N]{};
	int* dev_a{}, * dev_b{}, * dev_c{};
	// Alokacja pamięci na GPU
	HANDLE_ERROR(hipMalloc((void**)& dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)& dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)& dev_c, N * sizeof(int)));
	//Zapełnienie tablic a i b na CPU
	for (int i = 0; i < N; i++) 
	{
		a[i] = i;
		b[i] = i + i;
	}
	//Kopiowanie tablic a i b do GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
	addition<<<N, 1>>>(dev_a, dev_b, dev_c);
	// Kopiowanie tablicy c z GPU do CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
	// Wyświetlenie wyniku
	for (int i = 0; i < N; i++)
	{
		printf_s("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	// Zwolnienie pamięci alokowanej na GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	system("pause");
	return 0;
}

__global__ void addition(__int32* array_1, __int32* array_2, __int32* array_3)
{
	int id = blockIdx.x; // Działanie na danych znajdujących się pod tym indeksem
	if (id < N)
	{
		//add *100000 or more to get an error execution
		array_3[id] = (array_1[id] + array_2[id]);
		//array_3[id] = (array_1[id*10000000] + array_2[id]);
	}	
	//return __global__ void();
	/*
		It means that when we are invoking this function (less than N - 1 off course,
		cause our array size is N) we have to call blocks of GPU's contained in GRID.
		id = blockIdx.x is just a number of block currently (in easy way, index) we 
		working on
	*/
}
