#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"
#include "..//..//common/cpu_bitmap.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

using namespace std;

////////////////////////////////////////////////////
//GPU

#define DIM 100
#define OBJECT_SIZE (DIM/2)

class _Point
{
private:
	/*
		ZMIENNE PRIVATE
	*/
	float x;
	float y;
	//////////////////////////////////////////////////////////////////////////////
public:
	/*
		KONSTRUKTORY PUBLIC
	*/
	_Point() = delete;
	__device__ _Point(const float x, const float y);
	//////////////////////////////////////////////////////////////////////////////
	/*
		FUNKCJE PUBLIC
	*/

	//////////////////////////////////////////////////////////////////////////////
	/*
		SETTERY PUBLIC
	*/
	__device__ void Set_X(const float* new_X);
	__device__ void Set_Y(const float* new_Y);
	//////////////////////////////////////////////////////////////////////////////
	/*
		GETTERY PUBLIC
	*/
	__device__ constexpr float Get_X() const;
	__device__ constexpr float Get_Y() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		OPERATORY PUBLIC
	*/
	//JEDNOARGUMENTOWE
	__device__ _Point& operator+(const _Point& Object);
	__device__ _Point& operator-(const _Point& Object);
	__device__ _Point& operator*(const _Point& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		DESTRUKTOR
	*/
	__device__ ~_Point();		//wirtualny destruktor
	//////////////////////////////////////////////////////////////////////////////
};

__global__ void Kernel(unsigned char* ptr);
__device__ int Create_Square(int x, int y);

int main(int argc, char* argv[])
{
	CPUBitmap bitmap(DIM, DIM);
	unsigned char* dev_bitmap{};

	HANDLE_ERROR(hipMalloc((void**)& dev_bitmap, bitmap.image_size()));
	dim3 grid(DIM, DIM);

	Kernel<<<grid, 1 >>>(dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	bitmap.display_and_exit();
	HANDLE_ERROR(hipFree(dev_bitmap));

	system("pause");
	return 0;
}

// CLASS POINT CASE

__device__ _Point::_Point(const float x, const float y):
	x(x),
	y(y)
{

}

__device__ void _Point::Set_X(const float* new_X)
{
	this->x = (*new_X);
}

__device__ void _Point::Set_Y(const float* new_Y)
{
	this->y = (*new_Y);
}

__device__ constexpr float _Point::Get_X() const
{
	return this->x;
}

__device__ constexpr float _Point::Get_Y() const
{
	return this->y;
}

__device__ _Point& _Point::operator+(const _Point& Object)
{
	this->x += Object.x;
	this->y += Object.y;
	return *this;
}

__device__ _Point& _Point::operator-(const _Point& Object)
{
	this->x -= Object.x;
	this->y -= Object.y;
	return *this;
}

__device__ _Point& _Point::operator*(const _Point& Object)
{
	this->x *= Object.x;
	this->y *= Object.y;
	return *this;
}

_Point::~_Point()
{
	this->x = 0;
	this->y = 0;
}

////////////////////////////////////////////////////

__global__ void Kernel(unsigned char* ptr)
{
	// Odwzorowanie z blockldx na współrzędne piksela
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	// Obliczenie wartości dla tego punktu
	int my_value = Create_Square(x, y);
	ptr[offset * 4 + 0] = 255 * my_value;
	ptr[offset * 4 + 1] = 0;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}

__device__ int Create_Square(int x, int y)
{
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	_Point Object(jx, jy);

	for (size_t i = 0; i < 200; i++)
	{
		if (Object.Get_X() > OBJECT_SIZE)
		{
			return 0;
		}
	}
	return 1;
}