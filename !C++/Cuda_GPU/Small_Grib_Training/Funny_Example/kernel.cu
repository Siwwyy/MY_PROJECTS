#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "..//..//common/book.h"
#include "..//..//common/cpu_bitmap.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

/*
	ON GPU
*/
//This program wil get results of vary counts on complex numbers

//Zn+1= Zn^2+C

//#define DIM 20

using namespace std;

////////////////////////////////////////////////////
//CPU

//struct hipComplex 
//{
//	float r;
//	float i;
//	hipComplex(float a, float b) : r(a), i(b) {}
//	~hipComplex() { r = 0.0; i = 0.0f; }
//
//	float magnitude2(void)
//	{
//		return r * r + i * i;
//	}
//
//	hipComplex operator*(const hipComplex& a)
//	{
//		return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
//	}
//
//	hipComplex operator+(const hipComplex& a)
//	{
//		return hipComplex(r + a.r, i + a.i);
//	}
//
//};
//

//void kernel(unsigned char* ptr);
//int julia(int x, int y);
//
//int main(int argc, char* argv[])
//{
//	CPUBitmap bitmap(DIM, DIM);
//	unsigned char* ptr = bitmap.get_ptr();
//
//	kernel(ptr);
//		
//	system("pause");
//	return 0;
//}
//
//void kernel(unsigned char * ptr)
//{
//	for (int y = 0; y < DIM; y++)
//	{
//		for (int x = 0; x < DIM; x++)
//		{
//			int offset = x + y * DIM;
//			int juliaValue = julia(x, y);
//			ptr[offset * 4 + 0] = 255 * juliaValue;
//			ptr[offset * 4 + 1] = 0;
//			ptr[offset * 4 + 2] = 0;
//			ptr[offset * 4 + 3] = 255;
//		}
//	}
//}
//
//int julia(int x, int y)
//{
//	const float scale = 1.5;
//	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
//	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
//	hipComplex c(-0.8, 0.156);
//	hipComplex a(jx, jy);
//	int i = 0;
//	for (i = 0; i < 200; i++)
//	{
//		a = a * a + c;
//		if (a.magnitude2() > 1000)
//		{
//			return 0;
//		}
//	}
//	return 1;
//}

////////////////////////////////////////////////////
//GPU

#define DIM 1000

struct hipComplex
{
	float r;
	float i;
	__device__ hipComplex(float a, float b) : r(a), i(b) {}
	__device__ ~hipComplex() { r = 0.0; i = 0.0f; }

	__device__ float magnitude2(void)
	{
		return r * r + i * i;
	}

	__device__ hipComplex operator*(const hipComplex& a)
	{
		return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
	}

	__device__ hipComplex operator+(const hipComplex& a)
	{
		return hipComplex(r + a.r, i + a.i);
	}
};
	

__global__ void kernel(unsigned char * ptr);
__device__ int julia(int x, int y);

int main(int argc, char* argv[])
{
	CPUBitmap bitmap(DIM, DIM);
	unsigned char* dev_bitmap{};

	HANDLE_ERROR(hipMalloc((void**)& dev_bitmap,bitmap.image_size()));
	dim3 grid(DIM, DIM);


	kernel<<<grid, 1>>>(dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(),dev_bitmap,bitmap.image_size(),hipMemcpyDeviceToHost));

	bitmap.display_and_exit();
	HANDLE_ERROR(hipFree(dev_bitmap));

	system("pause");
	return 0;
}

__global__ void kernel(unsigned char* ptr)
{
	// Odwzorowanie z blockldx na współrzędne piksela
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;
	// Obliczenie wartości dla tego punktu
	int juliaValue = julia(x, y);
	ptr[offset * 4 + 0] = 255 * juliaValue;
	ptr[offset * 4 + 1] = 0;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}

__device__ int julia(int x, int y)
{
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);
	int i = 0;
	for (i = 0; i < 200; i++)
	{
		a = a * a + c;
		if (a.magnitude2() > 1000)
		{
			return 0;
		}
	}
	return 1;
}
